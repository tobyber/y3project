
#include "GL\glew.h"
#include "GL\wglew.h"
#pragma comment(lib, "glew32.lib")
//--------------

#include "glm\glm.hpp"
#include "glm\gtc\matrix_transform.hpp"
#include "glm\gtc\type_ptr.hpp"
#include "glm\gtc\matrix_inverse.hpp"

#include "GL\freeglut.h"

#include "hip/hip_runtime.h"
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#include ""

#include "shaders\Shader.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <iostream>
#include "vec3gpu.h"
#include "Ray.h"


	__device__ Ray::Ray()
	{
		this->dir = vec3gpu();
		this->origin = vec3gpu();
	}

	__device__ Ray::Ray(vec3gpu origin, vec3gpu direction)
	{
		this->dir = direction;
		this->origin = origin;


	}
	






