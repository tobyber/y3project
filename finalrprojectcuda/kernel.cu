#include "hip/hip_runtime.h"
﻿

#include "GL\glew.h"
#include "GL\wglew.h"
#pragma comment(lib, "glew32.lib")
//--------------

#include "glm\glm.hpp"
#include "glm\gtc\matrix_transform.hpp"
#include "glm\gtc\type_ptr.hpp"
#include "glm\gtc\matrix_inverse.hpp"

#include "GL\freeglut.h"

#include "hip/hip_runtime.h"
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#include ""

#include "shaders\Shader.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <iostream>
#include "vec3gpu.h"
#include "Ray.h"
#include "Sphere.h"
#include "World.h"


__device__ void renderRay(float4* pos, vec3gpu cameraPos, float x, float y, float width, float height, float left, float right, float top, float bottom);




__global__ void addKernel(float4* pos, int screen_width, int screen_height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y > screen_height || x > screen_width) return;
	

	//Ray r(	vec3(0, 0, 0), vec3(x / (float)screen_width, y / (float)screen_height, -1.0)	);
	//render_ray(pos, vec3(0,0,0), (float)x, (float)y, (float)screen_width, (float)screen_height, -1.0f, 1.0f, 1.0f, -1.0f);
	
	//Ray r();
	vec3gpu thing();
	World w;
	Sphere s(vec3gpu(0, 1, 3), 1.0, make_float4(1.0, 1.0, 0.0, 1.0));
	w.AddSphere(s);



	Ray r(pos, vec3gpu(0, 0, 0), (float)x, (float)y, (float)screen_width, (float)screen_height, -1.0f, 1.0f, 1.0f, -1.0f);

	float4 col = w.hitSpheres(r, float(x), float(y));
	pos[y * screen_width + x] = col;



	//pos[y*screen_width+x] = make_float4(r_col, g_col, z, z);
	//float test = 0.8f;
	//float r_col = sin(time / (float)screen_width);
	//float g_col = sin(time / (float)screen_height);
	//float z = 1.0;
	//pos[y*screen_width+x] = make_float4(r_col, g_col, z, z);
	
}



int prevTime = 0;
int window_width = 480;
int window_height = 480;


size_t num_bytes = (window_width * window_height) * 4 * sizeof(float);
CShader* myShader;
hipDeviceProp_t deviceProp;
GLuint vao;
GLuint vbo;
GLuint tex, buff;


hipGraphicsResource* cuda_tex_resource;

float vertices[] = {
    
    -1.0f, -1.0f, 0.0f,   0.0f, 0.0f,  // Bottom-left
    -1.0f,  1.0f, 0.0f,   0.0f, 1.0f,  // Top-left
     1.0f,  1.0f, 0.0f,   1.0f, 1.0f,  // Top-right

    -1.0f, -1.0f, 0.0f,   0.0f, 0.0f,  // Bottom-left
     1.0f,  1.0f, 0.0f,   1.0f, 1.0f,  // Top-right
     1.0f, -1.0f, 0.0f,   1.0f, 0.0f   // Bottom-right
};





void display();				//called in winmain to draw everything to the screen
//void reshape(int width, int height);				//called when the window is resized
void init();				//called in winmain when the program starts.
//void processKeys();         //called in winmain to process keyboard input
void idle();		//idle function


//https://github.com/Forceflow/cuda2GLcore/blob/main/src/main.cpp - memcpy into the texture instead?


//__device__ vec3gpu hitSphere(vec3gpu origin, vec3gpu dir, vec3gpu Center, float radius);



__device__ void renderRay(float4* pos, vec3gpu cameraPos, float x, float y, float width, float height, float left, float right, float top, float bottom)
{

	
	//sphere intersection

	
	//pos[(int)y * (int)width + (int)x] = make_float4(col, 0.0, 0.0, 1.0);


}
void display()
{
	
	glClear(GL_COLOR_BUFFER_BIT);
	
	
	//generate texture data from cuda
	float4* data_ptr;

	
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&data_ptr, &num_bytes,
		cuda_tex_resource));
	dim3 block(16, 16, 1);
	dim3 grid(window_width / block.x, window_height / block.y, 1);
	addKernel<<<grid, block >>>(data_ptr, window_width, window_height);

	//float test = time_f / (float)window_width;
	//std::cout << test << std::endl;
	hipGraphicsUnmapResources(1, &cuda_tex_resource, 0);
	

	glUseProgram(myShader->GetProgramObjID());

	//bind texture and get the data from pixel unpack buffer
	glBindTexture(GL_TEXTURE_2D, buff);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, tex);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, window_width, window_height, 0, GL_RGBA, GL_FLOAT, 0);


	//send the texture data to the shader
	//glActiveTexture(GL_TEXTURE0);
	//GLuint sampler = glGetUniformLocation(myShader->GetProgramObjID(), "sampler2D");
	//glUniform1i(sampler, 0);
	//glEnableVertexAttribArray(1);
	//glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));


	//draw the triangles
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);
	GLuint vbo1;
	glGenBuffers(1, &vbo1);
	glBindBuffer(GL_ARRAY_BUFFER, vbo1);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	

	glActiveTexture(GL_TEXTURE0);
	GLuint sampler = glGetUniformLocation(myShader->GetProgramObjID(), "sampler2D");
	glUniform1i(sampler, 0);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));


	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindTexture(GL_TEXTURE_2D, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

//	if (time_f >= 460) time_f = 0;
	//time_f += 5.0;
	glFlush();
	glutSwapBuffers();
}


void init()
{


	//create shader
	myShader = new CShader();
	if (!myShader->CreateShaderProgram("basic", "shader/shader.vert", "shader/shader.frag"))
	{
		std::cout << "failed to load shader" << std::endl;
	}

	//gen buffers and textures for output
	glEnable(GL_TEXTURE_2D);
	glGenBuffers(1, &tex);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, tex);

	glBufferData(GL_PIXEL_UNPACK_BUFFER, num_bytes, 0, GL_STATIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_tex_resource, tex, cudaGraphicsMapFlagsWriteDiscard));

	glGenTextures(1, &buff);
	glBindTexture(GL_TEXTURE_2D, buff);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glBindTexture(GL_TEXTURE_2D, 0);

	glClearColor(1.0, 1.0, 0.0, 1.0);
		
	//glDisable(GL_DEPTH_TEST);
	glEnable(GL_TEXTURE_2D);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	//gluPerspective(60, window_width / window_height, 0.1, 1);
	glOrtho(-1.0,window_width,-1.0,window_height,-1.0,1.0);

	prevTime = glutGet(GLUT_ELAPSED_TIME);




}









void idle()
{

	int curTime = glutGet(GLUT_ELAPSED_TIME);

	int dt = curTime - prevTime;

	//std::cout << dt << std::endl;

	prevTime = curTime;

	glutPostRedisplay();

}

int main(int argc, char** argv)
{

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutInitWindowSize(window_width, window_height);
	glutInitWindowPosition(100, 100);
	glutCreateWindow("test");

	GLenum err = glewInit();
	if (GLEW_OK != err)
	{
		std::cout << " GLEW ERROR" << std::endl;
	}
	const GLubyte* renderer = glGetString(GL_RENDERER);
	std::cout << "OPENGL RENDERER: " << renderer << std::endl;

	hipGetDeviceProperties(&deviceProp,0);
	hipSetDevice(0);
	std::cout << "CUDA DEVICE: "<< deviceProp.name << std::endl;
	//glutReshapeFunc(reshape);
	//specify which function will be called to refresh the screen.
	glutDisplayFunc(display);

	glutIdleFunc(idle);

	//starts the main loop. Program loops and calls callback functions as appropriate.
	init();
	glutMainLoop();

	return 0;

}
