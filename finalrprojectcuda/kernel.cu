#include "hip/hip_runtime.h"
﻿

#include "GL\glew.h"
#include "GL\wglew.h"
#pragma comment(lib, "glew32.lib")
//--------------

#include "glm\glm.hpp"
#include "glm\gtc\matrix_transform.hpp"
#include "glm\gtc\type_ptr.hpp"
#include "glm\gtc\matrix_inverse.hpp"

#include "GL\freeglut.h"

#include "hip/hip_runtime.h"
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#include ""

#include "imgui/imgui.h"
#include "imgui/imgui_impl_glut.h"
#include "imgui/imgui_impl_opengl3.h"




#include "shaders\Shader.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <iostream>
#include "vec3gpu.h"
#include "Ray.h"	
#include "Sphere.h"
#include "World.h"
#include "Light.h"
#include "Model.h"









void handleKeys();





__device__ void renderRay(float4* pos, vec3gpu cameraPos, float x, float y, float width, float height, float left, float right, float top, float bottom);




__global__ void addKernel(float4* pos, int screen_width, int screen_height,vec3gpu* modelTris ,UINT32 modelTrisNo, vec3gpu cameraPos, vec3gpu cameraRightDir,vec3gpu gpucamLookAt)
{

	
	vec3gpu background[4] = {
		vec3gpu(0,0,-1),
		vec3gpu(-20, -50, 10),
		vec3gpu(20, -50, 10),
		vec3gpu(20, 50, 10),

	};


	vec3gpu background2[4] = {
		vec3gpu(0,0,-1),
		vec3gpu(20, -50, 10),
		vec3gpu(-20, -50, 10),
		vec3gpu(-20, 50, 10),

	};

	vec3gpu floor[4] = {
		vec3gpu(0,1.0,0),
		vec3gpu(-20, 10, -20),
		vec3gpu(20, 10, -20),
		vec3gpu(20, 10, 20),

	};



	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y > screen_height || x > screen_width) return;

	vec3gpu thing();
	World w;
	Light l1(vec3gpu(0, -1 , 0), make_float4(1.0, 1.0, 1.0, 1.0));
	Sphere s1(vec3gpu(0, 3, -7), 1.0, make_float4(1.0, 0.0, 0.0, 1.0),1);
	Sphere s2(vec3gpu(0, 0, -5), 1.0, make_float4(1.0, 0.0, 1.0, 1.0),1);


	w.AddSphere(&s2);
	w.AddSphere(&s1);

	w.AddLight(l1);
	//w.AddModel(modelTris, modelTrisNo);
	w.AddModel(background, 1,make_float4(1.0,1.0,1.0,1.0));
	w.AddModel(background2, 1, make_float4(1.0, 1.0, 1.0, 1.0));
	float4 col = make_float4(0.0, 0.0, 0.0, 1.0);
	
	float xPos = cameraPos.x;
	float yPos = cameraPos.y;
	float zPos = cameraPos.z;
	Ray r(cameraPos,cameraRightDir, (float)x, (float)y, (float)screen_width, (float)screen_height,cameraPos.x-1.0f,cameraPos.x+1.0f,cameraPos.y-1.0f, cameraPos.y+1.0f,cameraPos.z+1.0f,gpucamLookAt);
	


	vec3gpu hitPoint;
	vec3gpu hitNormal;

		
		int sphereIntersect = -1;
		float closestT = 10000;
		sphereIntersect = w.hitClosestSphere(r, closestT);
		float modelT = 0;
		int hitmodel = w.testIntersect(r, l1, hitPoint, hitNormal,modelT);
		if ((modelT < closestT))
		{

			if (hitmodel != -1)
			{
			



			if (w.isPointInWorldShadow(hitPoint, l1))
			{
				col = make_float4(0.0, 0.0, 0.0, 1.0);

			}
			else
			{
				float4 newCol = w.colourModel(hitPoint, hitNormal, hitmodel, l1);
				col.x = newCol.x ;
				col.y = newCol.y ;
				col.z = newCol.z;

			}
		}

		}

		else if (sphereIntersect != -1)
		{
			vec3gpu hitPoint2 = r.origin + (r.dir * closestT);

			//if glass do this,

			//check material; visual artifacts
			//col = w.getColourFromReflect(r, hitPoint2, sphereIntersect, l1, cameraPos);
			float4 newCol = w.Spheres[sphereIntersect]->colourSphere(hitPoint2, cameraPos, l1);
			//else normal colour
			col.x = newCol.x;
			col.y = newCol.y;
			col.z = newCol.z;

			//}

		} 
		else
		{
			//hits nothing
	
		}


		

		
	


	pos[y * screen_width + x] = col;

	


	
}


//glut vars
int prevTime = 0;
int window_width = 480;
int window_height = 480;
int f_count = 0;

//key press
bool wPressed = false;
bool aPressed = false;
bool sPressed = false;
bool dPressed = false;
float cameraSpeed = 0.01;


//mouse handles
float yaw = 90;
float pitch = 0;
int	lastMouseX = 240, lastMouseY = 240;
glm::vec3 cameraPos = glm::vec3(0.0, 0.0, 0.0);
glm::vec3 cameraLookAt = glm::vec3(0.0, 0.0, -1.0);
glm::vec3 cameraDirection = glm::vec3(0.0, 0.0, 0.0);
glm::vec3 rightDir;
vec3gpu gpuRightDir;
vec3gpu gpucamLookAt(0, 0, -1.0);
vec3gpu gpuUpDir;

bool LeftPressed = false;
float* camMatrix;
float* gpucamMatrix;








size_t num_bytes = (window_width * window_height) * 4 * sizeof(float);
CShader* myShader;
hipDeviceProp_t deviceProp;
GLuint vao;
GLuint vbo;
GLuint tex, buff;
Model m1;
vec3gpu *modelTrisGPU;
vec3gpu gpucameraPos;


hipGraphicsResource* cuda_tex_resource;

float vertices[] = {
    
    -1.0f, -1.0f, 0.0f,   0.0f, 0.0f,  // Bottom-left
    -1.0f,  1.0f, 0.0f,   0.0f, 1.0f,  // Top-left
     1.0f,  1.0f, 0.0f,   1.0f, 1.0f,  // Top-right

    -1.0f, -1.0f, 0.0f,   0.0f, 0.0f,  // Bottom-left
     1.0f,  1.0f, 0.0f,   1.0f, 1.0f,  // Top-right
     1.0f, -1.0f, 0.0f,   1.0f, 0.0f   // Bottom-right
};





void display();				//called in winmain to draw everything to the screen
//void reshape(int width, int height);				//called when the window is resized
void init();				//called in winmain when the program starts.
//void processKeys();         //called in winmain to process keyboard input
void idle();		//idle function


//https://github.com/Forceflow/cuda2GLcore/blob/main/src/main.cpp - memcpy into the texture instead?


//__device__ vec3gpu hitSphere(vec3gpu origin, vec3gpu dir, vec3gpu Center, float radius);



__device__ void renderRay(float4* pos, vec3gpu gpucameraPos, float x, float y, float width, float height, float left, float right, float top, float bottom)
{

	
	//sphere intersection

	
	//pos[(int)y * (int)width + (int)x] = make_float4(col, 0.0, 0.0, 1.0);


}
void display()
{
	
	glClear(GL_COLOR_BUFFER_BIT);
	
	
	//generate texture data from cuda
	float4* data_ptr;
	
	

	//get pointer to opengl buffer
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&data_ptr, &num_bytes,
		cuda_tex_resource));

	
	dim3 block(16, 16, 1);
	dim3 grid(window_width / block.x, window_height / block.y, 1);
	addKernel<<<grid, block >>>(data_ptr, window_width, window_height,modelTrisGPU,m1.NUMBER_OF_TRIANGLES,gpucameraPos,gpuRightDir,gpucamLookAt);
	
	
	hipGraphicsUnmapResources(1, &cuda_tex_resource, 0);
	

	glUseProgram(myShader->GetProgramObjID());

	//bind texture and get the data from pixel unpack buffer
	glBindTexture(GL_TEXTURE_2D, buff);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, tex);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, window_width, window_height, 0, GL_RGBA, GL_FLOAT, 0);


	//send the texture data to the shader
	//glActiveTexture(GL_TEXTURE0);
	//GLuint sampler = glGetUniformLocation(myShader->GetProgramObjID(), "sampler2D");
	//glUniform1i(sampler, 0);
	//glEnableVertexAttribArray(1);
	//glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));


	//draw the triangles
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);
	GLuint vbo1;
	glGenBuffers(1, &vbo1);
	glBindBuffer(GL_ARRAY_BUFFER, vbo1);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	

	glActiveTexture(GL_TEXTURE0);
	GLuint sampler = glGetUniformLocation(myShader->GetProgramObjID(), "sampler2D");
	glUniform1i(sampler, 0);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));


	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindTexture(GL_TEXTURE_2D, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

//	if (time_f >= 460) time_f = 0;
	//time_f += 5.0;
	glFlush();
	glutSwapBuffers();
}


void init()
{

		

	m1 = Model("ImageToStl.stl");
	
	modelTrisGPU = m1.copyTrisToGPU();

	/*
	for (int i = 0; i < m1.NUMBER_OF_TRIANGLES; i++)
	{
		std::cout << "Triangle: " << i << std::endl;
		std::cout << "Normal:" << m1.triangles[4 * i + 0].x << std::endl;
		std::cout << "Vert 1:" << m1.triangles[4 * i + 1].x << std::endl;
		std::cout << "Vert 2:" << m1.triangles[4 * i + 2].x << std::endl;
		std::cout << "Vert 3:" << m1.triangles[4 * i + 3].x << std::endl;
			


	}
	*/


	//create shader
	myShader = new CShader();
	if (!myShader->CreateShaderProgram("basic", "shader/shader.vert", "shader/shader.frag"))
	{
		std::cout << "failed to load shader" << std::endl;
	}








	//gen buffers and textures for output
	glEnable(GL_TEXTURE_2D);
	glGenBuffers(1, &tex);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, tex);

	glBufferData(GL_PIXEL_UNPACK_BUFFER, num_bytes, 0, GL_STATIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_tex_resource, tex, cudaGraphicsMapFlagsWriteDiscard));

	glGenTextures(1, &buff);
	glBindTexture(GL_TEXTURE_2D, buff);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glBindTexture(GL_TEXTURE_2D, 0);

	glClearColor(1.0, 1.0, 0.0, 1.0);
		
	//glDisable(GL_DEPTH_TEST);
	glEnable(GL_TEXTURE_2D);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	//gluPerspective(60, window_width / window_height, 0.1, 1);
	glOrtho(-1.0,window_width,-1.0,window_height,-1.0,1.0);

	prevTime = glutGet(GLUT_ELAPSED_TIME);




}









void idle()
{

	int curTime = glutGet(GLUT_ELAPSED_TIME);

	int dt = curTime - prevTime;
	f_count++;
	//std::cout << dt << ',';

	prevTime = curTime;

	
	handleKeys();


	glutPostRedisplay();

}





void keyDown(unsigned char key, int x, int y)
{
	switch (key) {
	case 'w':
		wPressed = true;
		break;
	case 'a':
		aPressed = true;
		break;
	case 's':
		sPressed = true;
		break;
	case 'd':
		dPressed = true;
		break;

	}
}

void keyUp(unsigned char key, int x, int y)
{
	switch (key) {
	case 'w':
		wPressed = false;
		break;
	case 'a':
		aPressed = false;
		break;
	case 's':
		sPressed = false;
		break;
	case 'd':
		dPressed = false;
		break;
	}


}



void mouseMove(int x, int y) {

	//Inspired by learnopgl's code: https://learnopengl.com/Getting-started/Camera
	const float sensitivity = 0.05f;
	float xdiff = (x - lastMouseX) * sensitivity;
	float ydiff = (lastMouseY - y) * sensitivity;


	//lastMouseX = x;
	//lastMouseY = y;


	yaw += xdiff;
	pitch += ydiff;

	//allow cursor to stay in centre of screen
	glutWarpPointer(window_width / 2, window_height / 2);




	if (pitch > 89.0f)
		pitch = 89.0f;
	if (pitch < -89.0f)
		pitch = -89.0f;

	glm::vec3 direction = glm::vec3(0, 0, 0);
	direction.x = cos(glm::radians(yaw)) * cos(glm::radians(pitch));
	direction.y = sin(glm::radians(pitch));
	direction.z = sin(glm::radians(yaw)) * cos(glm::radians(pitch));
	cameraLookAt = glm::normalize(direction);
		
	
	
	
	glm::mat4 temp = glm::lookAt(cameraPos, cameraPos + cameraLookAt, glm::vec3(0, 1, 0));

	//take the inverse to go cam -> world, transpose to get in row-major order.
	temp = glm::inverseTranspose(temp);
	
	camMatrix = (float*)glm::value_ptr(temp);
	
	
	hipMalloc((void**)&gpucamMatrix, sizeof(float)*16);
	hipMemcpy(gpucamMatrix, camMatrix, sizeof(float)*16, hipMemcpyHostToDevice);

	
	gpucamLookAt.x = cameraPos.x + cameraLookAt.x;
	gpucamLookAt.y = cameraPos.y + cameraLookAt.y;
	gpucamLookAt.z = cameraPos.z + cameraLookAt.z;


	rightDir = glm::normalize(glm::cross(cameraLookAt, glm::vec3(0, 1, 0)));
	gpuRightDir.x = rightDir.x;
	gpuRightDir.y = rightDir.y;
	gpuRightDir.z = rightDir.z;
}




void handleKeys()
{

	if (wPressed )
	{
		cameraPos -= ( cameraLookAt* cameraSpeed);
		glm::vec3 newPos = (cameraLookAt * cameraSpeed);
		gpucameraPos.x -= newPos.x;
		gpucameraPos.y -= newPos.y;
		gpucameraPos.z -= newPos.z;
	}

	if (sPressed )
	{
		cameraPos += (cameraLookAt * cameraSpeed);
		glm::vec3 newPos =  (cameraLookAt * cameraSpeed);
		gpucameraPos.x += newPos.x;
		gpucameraPos.y += newPos.y;
		gpucameraPos.z += newPos.z;
	}

	if (aPressed )
	{

		glm::vec3 cameraLeftMove = glm::normalize(glm::cross(cameraLookAt, glm::vec3(0, 1, 0)));


		glm::vec3 posDiff = (cameraLeftMove * cameraSpeed);
		cameraPos += posDiff;
		gpucameraPos.x += posDiff.x;
		gpucameraPos.y += posDiff.y;
		gpucameraPos.z += posDiff.z;
	}
	if (dPressed )
	{
		glm::vec3 cameraLeftMove = glm::normalize(glm::cross(cameraLookAt, glm::vec3(0, 1, 0)));
		glm::vec3 posDiff = (cameraLeftMove * cameraSpeed);
		cameraPos -= posDiff;
		gpucameraPos.x -= posDiff.x;
		gpucameraPos.y -= posDiff.y;
		gpucameraPos.z -= posDiff.z;
	}




}


int main(int argc, char** argv)
{

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutInitWindowSize(window_width, window_height);
	glutInitWindowPosition(100, 100);
	glutCreateWindow("test");

	GLenum err = glewInit();
	if (GLEW_OK != err)
	{
		std::cout << " GLEW ERROR" << std::endl;
	}
	const GLubyte* renderer = glGetString(GL_RENDERER);
	std::cout << "OPENGL RENDERER: " << renderer << std::endl;

	hipGetDeviceProperties(&deviceProp,0);
	hipSetDevice(0);
	std::cout << "CUDA DEVICE: "<< deviceProp.name << std::endl;
	//glutReshapeFunc(reshape);
	//specify which function will be called to refresh the screen.
	glutDisplayFunc(display);
	glutKeyboardFunc(keyDown);
	glutKeyboardUpFunc(keyUp);
	glutPassiveMotionFunc(mouseMove);
	glutIdleFunc(idle);


	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO();
	io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
	


	//ImGui::StyleColorsDark();
	// Setup Platform/Renderer backends
	ImGui_ImplGLUT_Init();
	ImGui_ImplOpenGL3_Init();
	ImGui_ImplGLUT_InstallFuncs();







	//starts the main loop. Program loops and calls callback functions as appropriate.
	init();
	glutMainLoop();
	



	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGLUT_Shutdown();
	ImGui::DestroyContext();



	return 0;

}
